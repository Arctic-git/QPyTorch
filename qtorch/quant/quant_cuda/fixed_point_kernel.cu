#include "hip/hip_runtime.h"
#include "quant_kernel.h"
#include "sim_helper.cu"

// quantize an array of real numbers into fixed point with word length [wl] and [fl] fractional bits
// 2**-[sigma] is the smallest unit of the fixed point representation. Stochastic Rounding with r.
__global__ void fixed_point_quantize_kernel_stochastic(float* __restrict__ a,
                                                       float* __restrict__ r,
                                                       float* o, int size,
                                                       int sigma, bool clamp,
                                                       float t_min, float t_max) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    o[index] = round(a[index], r[index], sigma);
    if (clamp) {
      o[index] = clamp_helper(o[index], t_min, t_max);
    }
  }
}

// quantize an array of real numbers into fixed point with word length [wl] and [fl] fractional bits
// 2**-[sigma] is the smallest unit of the fixed point representation. Nearest Neighbor Rounding.
__global__ void fixed_point_quantize_kernel_nearest(float* __restrict__ a,
                                                    float* o, int size,
                                                    int sigma, bool clamp,
                                                    float t_min, float t_max) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    o[index] = round(a[index], 0.5, sigma);
    if (clamp) {
      o[index] = clamp_helper(o[index], t_min, t_max);
    }
  }
}