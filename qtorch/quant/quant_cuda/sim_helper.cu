#include "hip/hip_runtime.h"
#include "quant_kernel.h"

__device__ __forceinline__ float round_helper(float a, float r) {
  return floor(a+r);
}

template <typename T>
__device__ __forceinline__ T clamp_helper(T a, T min, T max) {
  if (a > max) return max;
  else if (a < min) return min;
  else return a;
}

__device__ __forceinline__ float round(float a, float r, int sigma) {
  a = ldexp(a, -sigma); 
  a = round_helper(a, r);
  a = ldexp(a, sigma);
  return a;
}