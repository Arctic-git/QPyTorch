
#include <hip/hip_runtime.h>
#define FLOAT_TO_BITS(x) (*reinterpret_cast<unsigned int*>(x))
#define BITS_TO_FLOAT(x) (*reinterpret_cast<float*>(x))

__device__ __forceinline__ unsigned int extract_exponent(float *a) {
  unsigned int temp = *(reinterpret_cast<unsigned int*>(a));
  temp = (temp << 1 >> 24); // single preciision, 1 sign bit, 23 mantissa bits
  return temp-127+1; // exponent offset and virtual bit
}

__device__ __forceinline__ unsigned int round_bitwise_stochastic(unsigned int target,
                                                                 unsigned int rand_prob,
                                                                 int man_bits) {
    unsigned int mask = (1 << (23-man_bits)) - 1;
    unsigned int add_r = target+(rand_prob & mask);
    unsigned int quantized = add_r & ~mask;
    return quantized;
}

__device__ __forceinline__ unsigned int round_bitwise_nearest(unsigned int target,
                                                              int man_bits) {
    unsigned int mask = (1 << (23-man_bits)) - 1;
    unsigned int rand_prob = 1 << (23-man_bits-1);
    unsigned int add_r = target+rand_prob;
    unsigned int quantized = add_r & ~mask;
    return quantized;
}

__device__ __forceinline__ unsigned int clip_exponent(int exp_bits, int man_bits,
                                                      unsigned int old_num,
                                                      unsigned int quantized_num) {
  // int offset = 32-9-man_bits; // float length minus sign bit and exponent bit add 1 virtual bit
  // unsigned int quantized_exponent_store = quantized_num << 1 >> 1 >> 23; // 1 sign bit, 23 mantissa bits
  // int quantized_exponent_real = (int) quantized_exponent_store - 126;
  // unsigned int max_exponent = (unsigned int) 1 << exp_bits;
  // if (quantized_exponent > max_exponent) {
  //   unsigned int max_man = (unsigned int ) -1 << (32-wl) >> 9; // 1 sign bit, 8 exponent bits
  //   unsigned int max_num = (max_exponent << 23) | max_man;
  //   unsigned int old_sign = old_num >> 31 << 31;
  //   quantized_num = old_sign | max_num;
  // }
  return quantized_num;
}