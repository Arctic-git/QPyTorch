#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <ATen/ATen.h>
#include "quant_kernel.h"

using namespace at;

__device__ __forceinline__ float stochastic_round_helper(float a, float r) {
  return floor(a+r);
}

template <typename T>
__device__ __forceinline__ T clamp_helper(T a, T min, T max) {
  if (a > max) return max;
  else if (a < min) return min;
  else return a;
}

__device__ __forceinline__ float stochastic_round(float a, float r, int sigma) {
  a = ldexp(a, -sigma); 
  a = stochastic_round_helper(a, r);
  a = ldexp(a, sigma);
  return a;
}

// __global__ void fixed_point_quantize_inplace_kernel(float *a,  float* __restrict__ r, int size,
//                                                     int sigma, float t_min, float t_max) {
//   int index = blockIdx.x * blockDim.x + threadIdx.x;
//   if (index < size) {
//     a[index] = stochastic_round(a[index], r[index], sigma);
//     a[index] = clamp_helper(a[index], t_min, t_max);
//   }
// }

// quantize an array of real numbers into fixed point with word length [wl] and [fl] fractional bits
// 2**-[sigma] is the smallest unit of the fixed point representation
__global__ void fixed_point_quantize_copy_kernel(float* __restrict__ a,
                                                 float* __restrict__ r,
                                                 float* o, int size, int sigma,
                                                 float t_min, float t_max) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    o[index] = stochastic_round(a[index], r[index], sigma);
    o[index] = clamp_helper(o[index], t_min, t_max);
  }
}

// __global__ void block_quantize_copy_kernel(float* __restrict__ a,
//                                            float* __restrict__ r,
//                                            float* o, int size, int wl,
//                                            short *exponent) {
//   int index = blockIdx.x * blockDim.x + threadIdx.x;
//   if (index < size) {
//     int sigma = (int) exponent[0]-(wl-1);
//     o[index] = stochastic_round(a[index], r[index], sigma);
//   }
// }

__device__ __forceinline__ short extract_exponent(float *a) {
  unsigned int temp = *(reinterpret_cast<unsigned int*>(a));
  temp = (short) (temp << 1 >> 24); // single preciision, 1 sign bit, 23 mantissa bits
  return temp-127+1; // exponent offset and virtual bit
}

// quantize an array of real number into block floating point
// each number has word length [wl] and [max_entry] is the maximum number
// in array
__global__ void block_quantize_copy_aten_kernel(float* __restrict__ a,
                                                float* __restrict__ r,
                                                float* o, int size, int wl,
                                                float *max_entry) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    short exponent = extract_exponent(max_entry);
    int sigma = (int) exponent-(wl-1);
    o[index] = stochastic_round(a[index], r[index], sigma);
  }
}

__global__ void float_kernel(float* __restrict__ a,
                                  float* __restrict__ r,
                                  float* o, int size,
                                  int exp_bits,
                                  int man_bits ) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    int exponent = (int) extract_exponent(a[index]);
    int max_exponent = (1 << (exp_bits - 1)) -1;
    int min_exponent = 1 << (exp_bits - 1);
    exponent = clamp_helper(exponent, max_exponent, min_exponent);
    int sigma = exponent - (man_bits-1);
    o[index] = stochastic_round(a[index], r[index], sigma);
  }
}

int64_t calc_size(Tensor a) {
  auto dim = a.dim();
  int64_t size = 1;
  for (int i=0; i<dim; i++) size *=a.size(i);
  return size;
}

Tensor float_quantize_cuda(Tensor a, Tensor r, int man_bits, int exp_bits) {
  // use external random number right now
  auto o = at::zeros_like(a);
  int64_t size = calc_size(a);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  float_kernel<<<blockNums, blockSize>>>(a.data<float>(),
                                         r.data<float>(),
                                         o.data<float>(),
                                         size,
                                         man_bits,
                                         exp_bits);
  return o;
}

Tensor fixed_point_quantize_cuda(Tensor a, Tensor r, int wl, int fl) {
  // use external random number right now
  auto o = at::zeros_like(a);
  int64_t size = calc_size(a);
  int sigma = -fl;
  float t_min = -ldexp(1.0, wl-fl-1);
  float t_max = -t_min-sigma;
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  fixed_point_quantize_copy_kernel<<<blockNums, blockSize>>>(a.data<float>(),
                                                             r.data<float>(),
                                                             o.data<float>(),
                                                             size,
                                                             sigma,
                                                             t_min,
                                                             t_max);
  return o;
}

Tensor block_quantize_aten_cuda(Tensor a, Tensor r, int wl) {
  auto o = at::zeros_like(a);
  int64_t size = calc_size(a);

  Tensor max_entry = at::max(at::abs(a));
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  block_quantize_copy_aten_kernel<<<blockNums, blockSize>>>(a.data<float>(),
                                                            r.data<float>(),
                                                            o.data<float>(),
                                                            size,
                                                            wl,
                                                            max_entry.data<float>());
  return o;

}

Tensor block_quantize_cuda(Tensor a, Tensor r, int wl) {
  auto o = at::zeros_like(a);
  auto dim = a.dim();
  int64_t size = 1;
  for (int i=0; i<dim; i++) size *=a.size(i);

  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  short *temp; 
  hipMalloc(&temp, blockNums*sizeof(short));

  extract_max_exponent_kernel<<<blockNums, blockSize>>>(a.data<float>(),
                                                        temp,
                                                        size);
  reduce_max_exponent_kernel<<<1, 1024>>>(temp,
                                          temp,
                                          blockNums);

  block_quantize_copy_kernel<<<blockNums, blockSize>>>(a.data<float>(),
                                                       r.data<float>(),
                                                       o.data<float>(),
                                                       size,
                                                       wl,
                                                       temp);
  hipFree(temp);
  return o;
}