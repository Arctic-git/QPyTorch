#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <climits>
#include <math.h>
#include <hip/hip_runtime.h>


using namespace std;
#define cudaCheckError() {                                          \
  hipError_t e=hipGetLastError();                                  \
  if(e!=hipSuccess) {                                               \
  printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
  exit(0); \
  }                                                                  \
}

#define FULL_MASK (unsigned int) (-1)

__device__ __forceinline__ short extract_exponent(float *a) {
  unsigned int temp = *(reinterpret_cast<unsigned int*>(a));
  temp = (short) (temp << 1 >> 24);
  return temp-(127-1);
}

__inline__ __device__ short warpReduceMax(short val) {
  for (int i=warpSize/2; i > 0; i = i/2) {
    short thread_exponent = __shfl_down_sync(FULL_MASK, val, i);
    val = (val > thread_exponent) ? val : thread_exponent;
  }

  return val;
}

__inline__ __device__ short blockReduceMax(short val) {
  static __shared__ short sdata[32]; 
  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;

  val = warpReduceMax(val);

  if (lane==0) sdata[wid] = val;
  __syncthreads();

  val = (threadIdx.x < blockDim.x / warpSize) ? sdata[lane] : SHRT_MIN;

  if (wid==0) val = warpReduceMax(val);

  return val;
}

__global__ void extract_max_exponent_kernel(float *a, short *o, int size)
{
  short max_exponent = SHRT_MIN;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i=index; i < size; i+= blockDim.x*gridDim.x)  {
    short thread_exponent = extract_exponent(a+i);
    max_exponent = (max_exponent > thread_exponent) ? max_exponent : thread_exponent;
  }
  max_exponent = blockReduceMax(max_exponent);

  if (threadIdx.x == 0) o[blockIdx.x] = max_exponent;

}


__global__ void reduce_max_exponent_kernel(short *a, short *o, int size)
{
  short max_exponent = SHRT_MIN;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i=index; i < size; i+= blockDim.x*gridDim.x)  {
    short thread_exponent = a[i];
    max_exponent = (max_exponent > thread_exponent) ? max_exponent : thread_exponent;
  }
  max_exponent = blockReduceMax(max_exponent);
  if (threadIdx.x == 0) o[blockIdx.x] = max_exponent;
}

void extract_max_exponent(float *a, int size) {
  int blockSize = 1024;
  int blockNums = min((size + blockSize - 1) / blockSize, 1024);

  // extract_max_exponent_kernel<<<blockNums, blockSize>>>(a, o, size); 
  // hipDeviceSynchronize();
  // cout << "final block reduce\n";
  // for (int i=0; i<20; i++) {
  //   cout << o[i] << "\n";
  // }

  short *out;
  hipMalloc(&out, blockNums*sizeof(short));
  extract_max_exponent_kernel<<<blockNums, blockSize>>>(a, out, size);
  reduce_max_exponent_kernel<<<1, 1024>>>(out, out, blockNums); 
  hipFree(out);
}

int main(void)
{
  int N = 128*128*32*32;
  float *x;
  short *max_exp;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&max_exp, min(N, 1024)*sizeof(short));


  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = i;
  }

  int device = -1;
  hipGetDevice(&device);
  hipMemPrefetchAsync(x, N*sizeof(float), device, NULL);
  // Run kernel on 1M elements on the GPU
  extract_max_exponent(x, N);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)


  hipFree(x);
  hipFree(max_exp);
  
  return 0;
}