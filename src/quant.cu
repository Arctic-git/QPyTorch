#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <math.h>
#include <hip/hip_runtime.h>

__device__ __forceinline__ float stochastic_round_helper(float a, float r) {
  return floor(a+r);
}

__device__ __forceinline__ float clamp_helper(float a, float min, float max) {
  if (a > max) return max;
  else if (a < min) return min;
  else return a;
}

__device__ __forceinline__ float stochastic_round(float a, float r, float sigma) {
  a /= sigma; 
  a = stochastic_round_helper(a, r);
  a *= sigma;
  return a;
}

__global__ void fixed_point_quantize_inplace_cuda(float *a,  float* __restrict__ r, int size,
                                     float sigma, float t_min, float t_max) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    a[index] = stochastic_round(a[index], r[index], sigma);
    a[index] = clamp_helper(a[index], t_min, t_max);
  }
}

__global__ void fixed_point_quantize_copy_cuda(float* __restrict__ a,
                                                  float* __restrict__ r,
                                                  float* o, int size, float sigma,
                                                  float t_min, float t_max) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    o[index] = stochastic_round(a[index], r[index], sigma);
    o[index] = clamp_helper(o[index], t_min, t_max);
  }
}

void fixed_point_quantize_inplace(float *a, float *r, int size, int wl, int fl) {
  float sigma = pow(2.0, -fl);
  float t_min = -pow(2.0, wl-fl-1);
  float t_max = -t_min-sigma;
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;
  fixed_point_quantize_inplace_cuda<<<blockNums, blockSize>>>(a, r, size,
                                                      sigma, t_min, t_max);
}

void fixed_point_quantize_copy(float *a, float *r, float *o, int size, int wl, int fl) {
  float sigma = pow(2.0, -fl);
  float t_min = -pow(2.0, wl-fl-1);
  float t_max = -t_min-sigma;
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;
  fixed_point_quantize_copy_cuda<<<blockNums, blockSize>>>(a, r, o, size,
                                                      sigma, t_min, t_max);
}

void print_array(float *a, int size) {
  std::cout << "{";
  for (int i=0; i<size-1; i++) {
    std::cout << a[i] << ",";
  }
  std::cout << a[size-1];
  std::cout << "}" << "\n";
}


int main(void){
  int N = 1<<20;
  float *a, *r, *o; 
  hipMallocManaged(&a, N*sizeof(float));
  hipMallocManaged(&r, N*sizeof(float));
  hipMallocManaged(&o, N*sizeof(float));

  for (int i=0; i<N; i++) {
    a[i] = ((float) rand() / RAND_MAX);
    r[i] = ((float) rand() / RAND_MAX);
    o[i] = 0;
  }

  int device = -1;
  hipGetDevice(&device);
  hipMemPrefetchAsync(a, N*sizeof(float), device, NULL);
  hipMemPrefetchAsync(r, N*sizeof(float), device, NULL);
  hipMemPrefetchAsync(o, N*sizeof(float), device, NULL);

  // std::cout << "before quantize a: ";
  // print_array(a, N);
  // std::cout << "r: ";
  // print_array(r, N);

  fixed_point_quantize_copy(a, r, o, N, 3, 2);
  hipDeviceSynchronize();
  fixed_point_quantize_inplace(a, r, N, 3, 2);
  hipDeviceSynchronize();

  // std::cout << "after quantize a: ";
  // print_array(a,N);
  // std::cout << "after quantize o: ";
  // print_array(o,N);

  hipFree(a);
  hipFree(r);
  hipFree(o);
  return 0;

}