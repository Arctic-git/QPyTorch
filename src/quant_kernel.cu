#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <ATen/ATen.h>

using namespace at;
__device__ __forceinline__ float stochastic_round_helper(float a, float r) {
  return floor(a+r);
}

__device__ __forceinline__ float clamp_helper(float a, float min, float max) {
  if (a > max) return max;
  else if (a < min) return min;
  else return a;
}

__device__ __forceinline__ float stochastic_round(float a, float r, float sigma) {
  a /= sigma; 
  a = stochastic_round_helper(a, r);
  a *= sigma;
  return a;
}



__global__ void fixed_point_quantize_inplace_kernel(float *a,  float* __restrict__ r, int size,
                                     float sigma, float t_min, float t_max) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    a[index] = stochastic_round(a[index], r[index], sigma);
    a[index] = clamp_helper(a[index], t_min, t_max);
  }
}

__global__ void fixed_point_quantize_copy_kernel(float* __restrict__ a,
                                                  float* __restrict__ r,
                                                  float* o, int size, float sigma,
                                                  float t_min, float t_max) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    o[index] = stochastic_round(a[index], r[index], sigma);
    o[index] = clamp_helper(o[index], t_min, t_max);
  }
}

Tensor fixed_point_quantize_cuda(Tensor a, Tensor r, int wl, int fl) {
  // use external random number right now
  auto o = at::zeros_like(a);
  auto dim = a.dim();
  int64_t size = 1;
  for (int i=0; i<dim; i++) size *=a.size(i);
  float sigma = pow(2.0, -fl);
  float t_min = -pow(2.0, wl-fl-1);
  float t_max = -t_min-sigma;
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  fixed_point_quantize_copy_kernel<<<blockNums, blockSize>>>(a.data<float>(),
                                                           r.data<float>(),
                                                           o.data<float>(),
                                                           size,
                                                           sigma,
                                                           t_min,
                                                           t_max);
  return o;
}

Tensor block_quantize_cuda(Tensor a, Tensor r, int wl, int fl) {
  // use external random number right now
  auto o = at::zeros_like(a);
  auto dim = a.dim();
  int64_t size = 1;
  for (int i=0; i<dim; i++) size *=a.size(i);
  float sigma = pow(2.0, -fl);
  float t_min = -pow(2.0, wl-fl-1);
  float t_max = -t_min-sigma;
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  fixed_point_quantize_copy_kernel<<<blockNums, blockSize>>>(a.data<float>(),
                                                           r.data<float>(),
                                                           o.data<float>(),
                                                           size,
                                                           sigma,
                                                           t_min,
                                                           t_max);
  return o;
}